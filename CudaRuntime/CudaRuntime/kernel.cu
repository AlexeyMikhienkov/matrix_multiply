#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define SIZE 2048


__global__ void multiplyMatrices(double* matrixA, double* matrixB, double* matrixMult) {
    double value;

    int initialRow = blockIdx.x, countBlocks = gridDim.x;
    int initialColumn = threadIdx.x, countThread = blockDim.x;

    for (int i = initialRow; i < SIZE; i += countBlocks)
        for (int j = initialColumn; j < SIZE; j += countThread) {
            value = 0;

            for (int k = 0; k < SIZE; ++k)
                value += matrixA[i * SIZE + k] * matrixB[k * SIZE + j];

            matrixMult[i * SIZE + j] = value;
        }
}

int main() {
    double* matrixA, * matrixB, * matrixMult;
    int sizeInt = SIZE * SIZE * sizeof(double);

    hipEvent_t start, stop;
    float gpuTime;

    hipMallocManaged(&matrixA, sizeInt);
    hipMallocManaged(&matrixB, sizeInt);
    hipMallocManaged(&matrixMult, sizeInt);

    for (int i = 0; i < SIZE * SIZE; ++i)
        matrixA[i] = matrixB[i] = 2;

    int _blocks = 32, _threads = 1024;
    dim3 threads(_threads);
    dim3 blocks(_blocks);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    multiplyMatrices << <blocks, threads >> > (matrixA, matrixB, matrixMult);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("blocks = %i, count threads = %i, time = %f", _blocks, _threads, gpuTime);

    hipEventDestroy(start); hipEventDestroy(stop);
    hipFree(matrixA); hipFree(matrixB); hipFree(matrixMult);
}